#include <time.h>
#include <stdio.h>
#include <hip/hip_runtime.h>

#include "hipblas.h"
#include <pthread.h>
#define SIZE 256

// ==========================================================================
// timer stuff

struct my_timer_t {
  const char *name;
  struct timespec start;
  struct timespec most_recent_tick;
};

static void timespec_subtract(struct timespec *x, struct timespec *y) {
  if (x->tv_nsec < y->tv_nsec) {
    x->tv_sec -= 1;
    x->tv_nsec += 1000000000L;
  }
  x->tv_sec -= y->tv_sec;
  x->tv_nsec -= y->tv_nsec;
}

static void report_elapsed(
  const char *name, 
  const char *msg, 
  struct timespec *x, 
  struct timespec *y)
{
  struct timespec diff = *x;
  timespec_subtract(&diff, y);
  double secs = (double)diff.tv_sec + ((double)diff.tv_nsec / 1000000000.0);
  printf("tick: %s: %s: elapsed: %lf\n", name, msg, secs);
}

void timer_begin(struct my_timer_t *t, const char *name) {
  t->name = name;
  clock_gettime(CLOCK_MONOTONIC, &(t->start));
  t->most_recent_tick = t->start;
}

void timer_report_tick(struct my_timer_t *t, const char *msg) {
  struct timespec prev = t->most_recent_tick;
  clock_gettime(CLOCK_MONOTONIC, &(t->most_recent_tick));
  report_elapsed(t->name, msg, &(t->most_recent_tick), &prev);
}


// ==========================================================================
// dMM boilerplate


/* TODO: inputs and outputs for leaf DMM, dimension info, etc. */
struct dMMPackage {
  // struct futStuff *futStuff;  /* won't need this */

  /* need to be specialized for DMM */
  float * a;
  float * b;
  float * output;
  uint64_t inputLen;

  /* these should stay */
  bool finished;
  pthread_t friends;
};

/* TODO: call cublas */
void* asyncdMMFunc(void* rawArg) {
  struct my_timer_t t;
  timer_begin(&t, "asyncdMMFunc");

  struct dMMPackage *pack = (struct dMMPackage *)rawArg;

  float *device_a;
  float *device_b;
  float *device_output;

  uint64_t n = pack->inputLen;
  uint64_t bytes = n*n*sizeof(float);

  hipMalloc(&device_a, bytes);
  hipMemcpy(device_a, pack->a, bytes, hipMemcpyHostToDevice);

  hipMalloc(&device_b, bytes);
  hipMemcpy(device_b, pack->b, bytes, hipMemcpyHostToDevice);
  
  hipMalloc(&(device_output), bytes);
  hipMemcpy(device_output, pack->output, bytes, hipMemcpyHostToDevice);
  // timer_report_tick(&t, "--- memcpy to gpu");

  float alpha = 1.0;
  float beta = 1.0;
  hipblasHandle_t handle;
  hipblasCreate(&handle);  
  hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &alpha, device_a, n, device_b, n, &beta, device_output, n);
  hipblasDestroy(handle);
  // timer_report_tick(&t, "      cublasSgemm");

  hipMemcpy(pack->output, device_output, bytes, hipMemcpyDeviceToHost);
  hipFree(device_a);
  hipFree(device_b);
  hipFree(device_output);
  // timer_report_tick(&t, "  memcpy from gpu");

  pack->finished = true; /* VERY IMPORTANT! */
  return NULL;
}


/* TODO: build the package, but otherwise shouldn't need to change much. 
 *
 * (NOTE: futhark_new_... is essentially a memcpy, these need to be replaced
 *  with stuff for cublas)
 */
extern "C" struct dMMPackage * 
dMMSpawn(
  float * a,
  float * b,
  float * output,
  int64_t inputLen)
{
  // struct futhark_context *ctx = futStuff->ctx;
  struct dMMPackage *pack = (dMMPackage*)malloc(sizeof(struct dMMPackage));
  // pack->futStuff = futStuff;
  // pack->a = futhark_new_u8_1d(ctx, a, inputLen);

  pack->a = a;
  pack->b = b;
  pack->output = output;
  pack->inputLen = inputLen;
  pack->finished = false;

  if (0 != pthread_create(&(pack->friends), NULL, &asyncdMMFunc, pack)) {
    printf("ERROR: glue.c: futdMMSpawn: pthread_create failed\n");
    exit(1);
  }

  return pack;
}

extern "C" uint8_t dMMPoll(struct dMMPackage *pack) {
  return pack->finished ? 1 : 0;
}


/* TODO: memcpy from GPU back to pack->output
 *
 * (NOTE: futhark_values is equivalent of this memcpy. needs to be replaced) */
extern "C" void dMMFinish(
  struct dMMPackage * pack)
{
  if (0 != pthread_join(pack->friends, NULL)) {
    printf("ERROR: glue.c: pthread_join failed\n");
    exit(1);
  }

  free(pack);
}

// ==========================================================================

extern "C"
void * memcpyFloatsToGpu(float *data, int64_t len) {
  struct my_timer_t t;
  timer_begin(&t, "memcpyFloatsToGpu");

  float *p;
  hipMalloc(&p, len*sizeof(float));
  hipMemcpyAsync(p, data, len*sizeof(float), hipMemcpyHostToDevice);

  timer_report_tick(&t, "done");
  return p;
}

extern "C"
void synchronizeGpu() {
  hipDeviceSynchronize();
}

extern "C"
void freeFloatsOnGpu(void *devicePtr) {
  hipFree(devicePtr);
}


// ==========================================================================


struct fancy_dmm_package {
  float * a;  // on device
  int64_t aTop;
  int64_t aLeft;
  int64_t aRowskip;
  float * b;  // on device
  int64_t bTop;
  int64_t bLeft;
  int64_t bRowskip;
  float * c;  // on host
  int64_t cTop;
  int64_t cLeft;
  int64_t cRowskip;
  int64_t n;

  /* these should stay */
  bool finished;
  pthread_t friends;
};


// copy into dst[0..n*n)
__global__
void copy_square_block(
  float *dst,
  uint64_t n,
  float *src,
  uint64_t top,
  uint64_t left,
  uint64_t rowskip)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;

  int total = n*n;
  for (int i = tid; i < total; i += stride) {
    int row = i/n;
    int col = i%n;
    int srcIdx = (top + row) * rowskip + left + col;
    dst[i] = src[srcIdx];
  }
}


extern "C"
void* fancy_dmm_func(void* rawArg) {
  struct my_timer_t t;
  timer_begin(&t, "fancy_dmm_func");

  struct fancy_dmm_package *pack = (struct fancy_dmm_package *)rawArg;

  uint64_t n = pack->n;
  uint64_t rowbytes = n*sizeof(float);
  uint64_t bytes = n*rowbytes;


  float *device_a;
  // float *tmp_a = (float*)malloc(bytes);
  hipMalloc(&device_a, bytes);
  /*
  for (int64_t j = 0; j < n; j++) {
    float *host_start = pack->a + (pack->aTop + j) * pack->aRowskip + pack->aLeft;
    cudaMemcpyAsync(device_a + j*n, host_start, rowbytes, cudaMemcpyDeviceToDevice);
    // memcpy(tmp_a + j*n, host_start, rowbytes);
  }
  */
  // cudaMemcpy(device_a, tmp_a, bytes, cudaMemcpyHostToDevice);
  // free(tmp_a);


  float *device_b;
  // float *tmp_b = (float*)malloc(bytes);
  hipMalloc(&device_b, bytes);
  /*
  for (int64_t j = 0; j < n; j++) {
    float *host_start = pack->b + (pack->bTop + j) * pack->bRowskip + pack->bLeft;
    cudaMemcpyAsync(device_b + j*n, host_start, rowbytes, cudaMemcpyDeviceToDevice);
    // memcpy(tmp_b + j*n, host_start, rowbytes);
  }
  */
  // cudaMemcpy(device_b, tmp_b, bytes, cudaMemcpyHostToDevice);
  // free(tmp_b);


  float *device_c;
  // float *tmp_c = (float*)malloc(bytes);
  hipMalloc(&device_c, bytes);
  for (int64_t j = 0; j < n; j++) {
    float *host_start = pack->c + (pack->cTop + j) * pack->cRowskip + pack->cLeft;
    hipMemcpyAsync(device_c + j*n, host_start, rowbytes, hipMemcpyHostToDevice);
    // memcpy(tmp_c + j*n, host_start, rowbytes);
  }
  // cudaMemcpy(device_c, tmp_c, bytes, cudaMemcpyHostToDevice);
  // free(tmp_c);

  int GRID = ((n*n)+(SIZE-1))/SIZE;
  if(GRID == 0) {
    GRID = 1;
  }
  copy_square_block<<<GRID, SIZE>>>(device_a, n, pack->a, pack->aTop, pack->aLeft, pack->aRowskip);
  // cudaDeviceSynchronize();

  copy_square_block<<<GRID, SIZE>>>(device_b, n, pack->b, pack->bTop, pack->bLeft, pack->bRowskip);
  hipDeviceSynchronize();

  timer_report_tick(&t, "--- memcpy to gpu");

  float alpha = 1.0;
  float beta = 1.0;
  hipblasHandle_t handle;
  hipblasCreate(&handle);  
  hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &alpha, device_a, n, device_b, n, &beta, device_c, n);
  hipblasDestroy(handle);
  timer_report_tick(&t, "      hipblasSgemm");


  // float *tmp_c = (float*)malloc(bytes);
  // cudaMemcpy(tmp_c, device_c, bytes, cudaMemcpyDeviceToHost);

  for (int64_t j = 0; j < n; j++) {
    float *host_start = pack->c + (pack->cTop + j) * pack->cRowskip + pack->cLeft;
    // memcpy(host_start, tmp_c + j*n, rowbytes);
    hipMemcpyAsync(host_start, device_c + j*n, rowbytes, hipMemcpyDeviceToHost);
  }
  // free(tmp_c);
  hipDeviceSynchronize();

  hipFree(device_a);
  hipFree(device_b);
  hipFree(device_c);
  timer_report_tick(&t, "  memcpy from gpu");

  pack->finished = true; /* VERY IMPORTANT! */
  return NULL;
}


extern "C" struct fancy_dmm_package * 
fancy_dmm_spawn(
  float * a,     // on device
  int64_t aTop,
  int64_t aLeft,
  int64_t aRowskip,
  float * b,     // on device
  int64_t bTop,
  int64_t bLeft,
  int64_t bRowskip,
  float * c,     // on host
  int64_t cTop,
  int64_t cLeft,
  int64_t cRowskip,
  int64_t n)
{
  struct fancy_dmm_package *pack = (fancy_dmm_package*)malloc(sizeof(struct fancy_dmm_package));

  pack->a = a;
  pack->aTop = aTop;
  pack->aLeft = aLeft;
  pack->aRowskip = aRowskip;

  pack->b = b;
  pack->bTop = bTop;
  pack->bLeft = bLeft;
  pack->bRowskip = bRowskip;

  pack->c = c;
  pack->cTop = cTop;
  pack->cLeft = cLeft;
  pack->cRowskip = cRowskip;

  pack->n = n;

  pack->finished = false;
  if (0 != pthread_create(&(pack->friends), NULL, &fancy_dmm_func, pack)) {
    printf("ERROR: glue.c: futdMMSpawn: pthread_create failed\n");
    exit(1);
  }

  return pack;
}


extern "C" uint8_t fancy_dmm_poll(struct fancy_dmm_package *pack) {
  return pack->finished ? 1 : 0;
}


/* TODO: memcpy from GPU back to pack->output
 *
 * (NOTE: futhark_values is equivalent of this memcpy. needs to be replaced) */
extern "C" void fancy_dmm_finish(
  struct fancy_dmm_package * pack)
{
  if (0 != pthread_join(pack->friends, NULL)) {
    printf("ERROR: glue.c: pthread_join failed\n");
    exit(1);
  }

  free(pack);
}