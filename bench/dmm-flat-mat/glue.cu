#include <time.h>
#include <stdio.h>
#include <hip/hip_runtime.h>

#include "hipblas.h"
#include <pthread.h>
#include <error.h>

#define SIZE 256

#define ENABLE_TIMER_TICKS false

// ==========================================================================
// timer stuff

struct my_timer_t {
  const char *name;
  struct timespec start;
  struct timespec most_recent_tick;
};

static void timespec_subtract(struct timespec *x, struct timespec *y) {
  if (x->tv_nsec < y->tv_nsec) {
    x->tv_sec -= 1;
    x->tv_nsec += 1000000000L;
  }
  x->tv_sec -= y->tv_sec;
  x->tv_nsec -= y->tv_nsec;
}

static void report_elapsed(
  const char *name, 
  const char *msg, 
  struct timespec *x, 
  struct timespec *y)
{
  struct timespec diff = *x;
  timespec_subtract(&diff, y);
  double secs = (double)diff.tv_sec + ((double)diff.tv_nsec / 1000000000.0);
  printf("tick: %s: %s: elapsed: %lf\n", name, msg, secs);
  fflush(stdout);
}

void timer_begin(struct my_timer_t *t, const char *name) {
  t->name = name;
  clock_gettime(CLOCK_MONOTONIC, &(t->start));
  t->most_recent_tick = t->start;
}

void timer_report_tick(struct my_timer_t *t, const char *msg) {
  struct timespec prev = t->most_recent_tick;
  clock_gettime(CLOCK_MONOTONIC, &(t->most_recent_tick));
  if (ENABLE_TIMER_TICKS) {
    report_elapsed(t->name, msg, &(t->most_recent_tick), &prev);
  }
}


// ==========================================================================
// dMM boilerplate


/* TODO: inputs and outputs for leaf DMM, dimension info, etc. */
struct dMMPackage {
  // struct futStuff *futStuff;  /* won't need this */

  /* need to be specialized for DMM */
  float * a;
  float * b;
  float * output;
  uint64_t inputLen;

  /* these should stay */
  bool finished;
  pthread_t friends;
};


void set_cpu_affinity(int cpu) {
  cpu_set_t cpuset;
  pthread_t thread;

  thread = pthread_self();

  CPU_ZERO(&cpuset);
  CPU_SET(cpu, &cpuset);

  if (pthread_setaffinity_np(thread, sizeof cpuset, &cpuset) != 0) {
    printf("ERROR: glue.c: could not set affinity\n");
    exit(1);
  }
}


void* asyncdMMFunc(void* rawArg) {
  struct my_timer_t t;
  timer_begin(&t, "asyncdMMFunc");

  // set_cpu_affinity(31);

  struct dMMPackage *pack = (struct dMMPackage *)rawArg;

  float *device_a;
  float *device_b;
  float *device_output;

  uint64_t n = pack->inputLen;
  uint64_t bytes = n*n*sizeof(float);

  hipMalloc(&device_a, bytes);
  hipMemcpy(device_a, pack->a, bytes, hipMemcpyHostToDevice);

  hipMalloc(&device_b, bytes);
  hipMemcpy(device_b, pack->b, bytes, hipMemcpyHostToDevice);
  
  hipMalloc(&(device_output), bytes);
  hipMemcpy(device_output, pack->output, bytes, hipMemcpyHostToDevice);
  // timer_report_tick(&t, "--- memcpy to gpu");

  float alpha = 1.0;
  float beta = 1.0;
  hipblasHandle_t handle;
  hipblasCreate(&handle);  
  hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &alpha, device_a, n, device_b, n, &beta, device_output, n);
  hipblasDestroy(handle);
  // timer_report_tick(&t, "      cublasSgemm");

  hipMemcpy(pack->output, device_output, bytes, hipMemcpyDeviceToHost);
  hipFree(device_a);
  hipFree(device_b);
  hipFree(device_output);
  // timer_report_tick(&t, "  memcpy from gpu");

  __atomic_store_n(&(pack->finished), (bool)true, __ATOMIC_SEQ_CST); /* VERY IMPORTANT! */
  return NULL;
}

// ==========================================================================

extern "C"
void * memcpyFloatsToGpu(float *data, int64_t len) {
  struct my_timer_t t;
  timer_begin(&t, "memcpyFloatsToGpu");

  float *p;
  hipMalloc(&p, len*sizeof(float));
  hipMemcpyAsync(p, data, len*sizeof(float), hipMemcpyHostToDevice);

  timer_report_tick(&t, "done");
  return p;
}

extern "C"
void synchronizeGpu() {
  hipDeviceSynchronize();
}

extern "C"
void freeFloatsOnGpu(void *devicePtr) {
  hipFree(devicePtr);
}


// ==========================================================================


// copy into dst[0..n*n)
__global__
void copy_block(
  float *dst,
  uint64_t height,
  uint64_t width,
  float *src,
  uint64_t top,
  uint64_t left,
  uint64_t rowskip)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;

  int total = height*width;
  for (int i = tid; i < total; i += stride) {
    int row = i/width;
    int col = i%width;
    int srcIdx = (top + row) * rowskip + left + col;
    dst[i] = src[srcIdx];
  }
}


// ==========================================================================


struct fancy_dmm_package {
  float * a;  // on device
  int64_t aTop;
  int64_t aLeft;
  int64_t aRowskip;
  float * b;  // on device
  int64_t bTop;
  int64_t bLeft;
  int64_t bRowskip;
  float * c;  // on host
  int64_t cTop;
  int64_t cLeft;
  int64_t cRowskip;
  int64_t m;
  int64_t n;
  int64_t k;

  /* these should stay */
  bool finished;
  pthread_t friends;
};



extern "C"
void* fancy_dmm_func(void* rawArg) {
  struct my_timer_t t;
  timer_begin(&t, "fancy_dmm_func");

  struct fancy_dmm_package *pack = (struct fancy_dmm_package *)rawArg;

  uint64_t m = pack->m;
  uint64_t n = pack->n;
  uint64_t k = pack->k;

  uint64_t abytes = m*k*sizeof(float);
  uint64_t bbytes = k*n*sizeof(float);
  uint64_t cbytes = m*n*sizeof(float);

  float *device_a;
  float *device_b;
  float *device_c;
  hipMalloc(&device_a, abytes);
  hipMalloc(&device_b, bbytes);
  hipMalloc(&device_c, cbytes);

  int GRID = ((n*n)+(SIZE-1))/SIZE;
  if(GRID == 0) {
    GRID = 1;
  }
  copy_block<<<GRID, SIZE>>>(device_a, m, k, pack->a, pack->aTop, pack->aLeft, pack->aRowskip);
  // cudaDeviceSynchronize();

  copy_block<<<GRID, SIZE>>>(device_b, k, n, pack->b, pack->bTop, pack->bLeft, pack->bRowskip);
  hipDeviceSynchronize();

  timer_report_tick(&t, "--- memcpy to gpu");

  float alpha = 1.0;
  float beta = 0.0;
  hipblasHandle_t handle;
  hipblasCreate(&handle);  
  hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, device_a, m, device_b, k, &beta, device_c, m);
  hipblasDestroy(handle);
  timer_report_tick(&t, "      hipblasSgemm");


  hipMemcpy(pack->c, device_c, cbytes, hipMemcpyDeviceToHost);

  hipFree(device_a);
  hipFree(device_b);
  hipFree(device_c);
  timer_report_tick(&t, "  memcpy from gpu");

  __atomic_store_n(&(pack->finished), (bool)true, __ATOMIC_SEQ_CST); /* VERY IMPORTANT! */
  return NULL;
}


extern "C" struct fancy_dmm_package * 
fancy_dmm_spawn(
  float * a,     // on device
  int64_t aTop,
  int64_t aLeft,
  int64_t aRowskip,
  float * b,     // on device
  int64_t bTop,
  int64_t bLeft,
  int64_t bRowskip,
  float * c,     // on host
  int64_t cTop,
  int64_t cLeft,
  int64_t cRowskip,
  int64_t m,
  int64_t n,
  int64_t k)
{
  struct fancy_dmm_package *pack = (fancy_dmm_package*)malloc(sizeof(struct fancy_dmm_package));

  pack->a = a;
  pack->aTop = aTop;
  pack->aLeft = aLeft;
  pack->aRowskip = aRowskip;

  pack->b = b;
  pack->bTop = bTop;
  pack->bLeft = bLeft;
  pack->bRowskip = bRowskip;

  pack->c = c;
  pack->cTop = cTop;
  pack->cLeft = cLeft;
  pack->cRowskip = cRowskip;

  pack->m = m;
  pack->n = n;
  pack->k = k;

  pack->finished = false;

  fancy_dmm_func(pack);

  // if (0 != pthread_create(&(pack->friends), NULL, &fancy_dmm_func, pack)) {
  //   printf("ERROR: glue.c: futdMMSpawn: pthread_create failed\n");
  //   exit(1);
  // }

  return pack;
}


/* TODO: memcpy from GPU back to pack->output
 *
 * (NOTE: futhark_values is equivalent of this memcpy. needs to be replaced) */
extern "C" void fancy_dmm_finish(
  struct fancy_dmm_package * pack)
{
  // if (0 != pthread_join(pack->friends, NULL)) {
  //   printf("ERROR: glue.c: pthread_join failed\n");
  //   exit(1);
  // }

  free(pack);
}


// ===========================================================================


struct fancy_two_dmm_package {
  float * a;  // on device
  int64_t aTop1;
  int64_t aLeft1;
  int64_t aTop2;
  int64_t aLeft2;
  int64_t aRowskip;
  float * b;  // on device
  int64_t bTop1;
  int64_t bLeft1;
  int64_t bTop2;
  int64_t bLeft2;
  int64_t bRowskip;
  float * c;  // on host
  int64_t cTop;
  int64_t cLeft;
  int64_t cRowskip;
  int64_t n;

  /* these should stay */
  bool finished;
  pthread_t friends;
};



extern "C"
void* fancy_two_dmm_func(void* rawArg) {
  struct my_timer_t t;
  timer_begin(&t, "fancy_two_dmm_func");

  struct fancy_two_dmm_package *pack = (struct fancy_two_dmm_package *)rawArg;

  uint64_t n = pack->n;
  uint64_t rowbytes = n*sizeof(float);
  uint64_t bytes = n*rowbytes;


  float *device_c;
  hipMalloc(&device_c, bytes);
  // for (int64_t j = 0; j < n; j++) {
  //   float *host_start = pack->c + (pack->cTop + j) * pack->cRowskip + pack->cLeft;
  //   cudaMemcpyAsync(device_c + j*n, host_start, rowbytes, cudaMemcpyHostToDevice);
  // }

  // cudaDeviceSynchronize();
  // timer_report_tick(&t, "----- memcpy C to gpu");

  float *tmp_a;
  float *tmp_b;
  hipMalloc(&tmp_a, bytes);
  hipMalloc(&tmp_b, bytes);


  int GRID = ((n*n)+(SIZE-1))/SIZE;
  if(GRID == 0) {
    GRID = 1;
  }
  copy_block<<<GRID, SIZE>>>(tmp_a, n, n, pack->a, pack->aTop1, pack->aLeft1, pack->aRowskip);
  copy_block<<<GRID, SIZE>>>(tmp_b, n, n, pack->b, pack->bTop1, pack->bLeft1, pack->bRowskip);
  hipDeviceSynchronize();
  timer_report_tick(&t, "- memcpy A1,B1 on gpu");

  float alpha = 1.0;
  float beta = 0.0;
  hipblasHandle_t handle;
  hipblasCreate(&handle);  
  hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &alpha, tmp_a, n, tmp_b, n, &beta, device_c, n);
  hipblasDestroy(handle);
  timer_report_tick(&t, "   cublasSgemm(A1,B1)");
  

  copy_block<<<GRID, SIZE>>>(tmp_a, n, n, pack->a, pack->aTop2, pack->aLeft2, pack->aRowskip);
  copy_block<<<GRID, SIZE>>>(tmp_b, n, n, pack->b, pack->bTop2, pack->bLeft2, pack->bRowskip);
  hipDeviceSynchronize();
  timer_report_tick(&t, "  memcpy A2,B2 on gpu");


  beta = 1.0;
  hipblasCreate(&handle);  
  hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &alpha, tmp_a, n, tmp_b, n, &beta, device_c, n);
  hipblasDestroy(handle);
  timer_report_tick(&t, "   cublasSgemm(A2,B2)");

  // for (int64_t j = 0; j < n; j++) {
  //   float *host_start = pack->c + (pack->cTop + j) * pack->cRowskip + pack->cLeft;
  //   cudaMemcpyAsync(host_start, device_c + j*n, rowbytes, cudaMemcpyDeviceToHost);
  // }
  // cudaDeviceSynchronize();

  hipMemcpy(pack->c, device_c, bytes, hipMemcpyDeviceToHost);
  hipFree(tmp_a);
  hipFree(tmp_b);
  hipFree(device_c);
  timer_report_tick(&t, "    memcpy C from gpu");

  __atomic_store_n(&(pack->finished), (bool)true, __ATOMIC_SEQ_CST); /* VERY IMPORTANT! */
  return NULL;
}


extern "C" struct fancy_two_dmm_package * 
fancy_two_dmm_spawn(
  float * a,     // on device
  int64_t aTop1,
  int64_t aLeft1,
  int64_t aTop2,
  int64_t aLeft2,
  int64_t aRowskip,
  float * b,     // on device
  int64_t bTop1,
  int64_t bLeft1,
  int64_t bTop2,
  int64_t bLeft2,
  int64_t bRowskip,
  float * c,     // on host
  int64_t cTop,
  int64_t cLeft,
  int64_t cRowskip,
  int64_t n)
{
  struct fancy_two_dmm_package *pack = (fancy_two_dmm_package*)malloc(sizeof(struct fancy_two_dmm_package));

  pack->a = a;
  pack->aTop1 = aTop1;
  pack->aLeft1 = aLeft1;
  pack->aTop2 = aTop2;
  pack->aLeft2 = aLeft2;
  pack->aRowskip = aRowskip;

  pack->b = b;
  pack->bTop1 = bTop1;
  pack->bLeft1 = bLeft1;
  pack->bTop2 = bTop2;
  pack->bLeft2 = bLeft2;
  pack->bRowskip = bRowskip;

  pack->c = c;
  pack->cTop = cTop;
  pack->cLeft = cLeft;
  pack->cRowskip = cRowskip;

  pack->n = n;

  pack->finished = false;

  fancy_two_dmm_func(pack);

  // if (0 != pthread_create(&(pack->friends), NULL, &fancy_two_dmm_func, pack)) {
  //   printf("ERROR: glue.c: futdMMSpawn: pthread_create failed\n");
  //   exit(1);
  // }

  return pack;
}


// extern "C" uint8_t fancy_two_dmm_poll(struct fancy_two_dmm_package *pack) {
//   return pack->finished ? 1 : 0;
// }


extern "C" void fancy_two_dmm_finish(
  struct fancy_two_dmm_package * pack)
{
  // if (0 != pthread_join(pack->friends, NULL)) {
  //   printf("ERROR: glue.c: pthread_join failed\n");
  //   exit(1);
  // }

  free(pack);
}