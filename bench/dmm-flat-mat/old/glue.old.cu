#include <time.h>
#include <stdio.h>
#include <hip/hip_runtime.h>

#include "hipblas.h"
#include <pthread.h>
#include <error.h>

#define SIZE 256

#define ENABLE_TIMER_TICKS false

// ==========================================================================
// timer stuff

struct my_timer_t {
  const char *name;
  struct timespec start;
  struct timespec most_recent_tick;
};

static void timespec_subtract(struct timespec *x, struct timespec *y) {
  if (x->tv_nsec < y->tv_nsec) {
    x->tv_sec -= 1;
    x->tv_nsec += 1000000000L;
  }
  x->tv_sec -= y->tv_sec;
  x->tv_nsec -= y->tv_nsec;
}

static void report_elapsed(
  const char *name, 
  const char *msg, 
  struct timespec *x, 
  struct timespec *y)
{
  struct timespec diff = *x;
  timespec_subtract(&diff, y);
  double secs = (double)diff.tv_sec + ((double)diff.tv_nsec / 1000000000.0);
  printf("tick: %s: %s: elapsed: %lf\n", name, msg, secs);
  fflush(stdout);
}

void timer_begin(struct my_timer_t *t, const char *name) {
  t->name = name;
  clock_gettime(CLOCK_MONOTONIC, &(t->start));
  t->most_recent_tick = t->start;
}

void timer_report_tick(struct my_timer_t *t, const char *msg) {
  struct timespec prev = t->most_recent_tick;
  clock_gettime(CLOCK_MONOTONIC, &(t->most_recent_tick));
  if (ENABLE_TIMER_TICKS) {
    report_elapsed(t->name, msg, &(t->most_recent_tick), &prev);
  }
}

void set_cpu_affinity(int cpu) {
  cpu_set_t cpuset;
  pthread_t thread;

  thread = pthread_self();

  CPU_ZERO(&cpuset);
  CPU_SET(cpu, &cpuset);

  if (pthread_setaffinity_np(thread, sizeof cpuset, &cpuset) != 0) {
    printf("ERROR: glue.c: could not set affinity\n");
    exit(1);
  }
}

#define GPU_ID_BUFFER_SIZE 16

// static long parse_long(const char *str)
// {
//   errno = 0;
//   char *temp;
//   long val = strtol(str, &temp, 0);

//   if (temp == str || *temp != '\0' ||
//       ((val == LONG_MIN || val == LONG_MAX) && errno == ERANGE))
//   {
//     fprintf(stderr, "Could not convert '%s' to long and leftover string is: '%s'\n",
//             str, temp);
//     exit(1);
//   }
//   return val;
// }

int parse_cuda_device(
  char * gpu_id,
  int64_t gpu_id_str_len)
{
  char buf[GPU_ID_BUFFER_SIZE];
  if (gpu_id_str_len <= 0 || gpu_id_str_len > GPU_ID_BUFFER_SIZE || gpu_id[0] != '#') {
    printf("ERROR: glue.cu: parse_cuda_device: bad gpu_id\n");
    exit(1);
  }
  strncpy(&(buf[0]), gpu_id+1, gpu_id_str_len-1);
  buf[gpu_id_str_len-1] = '\0';
  char *end;
  long result = strtol(buf, &end, 10);
  if (end != buf + gpu_id_str_len-1) {
    printf("ERROR: glue.cu: parse_cuda_device: strtol failed\n");
    printf("buf contents: %s\n", buf);
    printf("result %ld\n", result);
    printf("buf %p gpu_id_str_len %ld end %p\n", (void*)buf, gpu_id_str_len, (void*)end);
    exit(1);
  }

  return (int)result;
}

void set_cuda_device(
  char * gpu_id,
  int64_t gpu_id_str_len)
{
  hipSetDevice(parse_cuda_device(gpu_id, gpu_id_str_len));
}

// ==========================================================================

extern "C"
void * memcpyFloatsToGpu(
  char * gpu_id,
  int64_t gpu_id_str_len,
  float *data,
  int64_t len)
{
  struct my_timer_t t;
  timer_begin(&t, "memcpyFloatsToGpu");

  set_cuda_device(gpu_id, gpu_id_str_len);

  float *p;
  hipMalloc(&p, len*sizeof(float));
  hipMemcpyAsync(p, data, len*sizeof(float), hipMemcpyHostToDevice);

  timer_report_tick(&t, "done");
  return p;
}

extern "C"
void synchronizeGpu(
  char * gpu_id,
  int64_t gpu_id_str_len)
{
  set_cuda_device(gpu_id, gpu_id_str_len);
  hipDeviceSynchronize();
}

extern "C"
void freeFloatsOnGpu(
  char * gpu_id,
  int64_t gpu_id_str_len,
  void *devicePtr)
{
  set_cuda_device(gpu_id, gpu_id_str_len);
  hipFree(devicePtr);
}


// ==========================================================================


// copy into dst[0..n*n)
__global__
void copy_block(
  float *dst,
  uint64_t height,
  uint64_t width,
  float *src,
  uint64_t top,
  uint64_t left,
  uint64_t rowskip)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;

  int total = height*width;
  for (int i = tid; i < total; i += stride) {
    int row = i/width;
    int col = i%width;
    int srcIdx = (top + row) * rowskip + left + col;
    dst[i] = src[srcIdx];
  }
}


// ==========================================================================


struct fancy_dmm_package {
  float * a;  // on device
  int64_t aTop;
  int64_t aLeft;
  int64_t aRowskip;
  float * b;  // on device
  int64_t bTop;
  int64_t bLeft;
  int64_t bRowskip;
  float * c;  // on host
  int64_t cTop;
  int64_t cLeft;
  int64_t cRowskip;
  int64_t m;
  int64_t n;
  int64_t k;

  /* these should stay */
  bool finished;
  pthread_t friends;
};



extern "C"
void* fancy_dmm_func(void* rawArg) {
  struct my_timer_t t;
  timer_begin(&t, "fancy_dmm_func");

  struct fancy_dmm_package *pack = (struct fancy_dmm_package *)rawArg;

  uint64_t m = pack->m;
  uint64_t n = pack->n;
  uint64_t k = pack->k;

  uint64_t abytes = m*k*sizeof(float);
  uint64_t bbytes = k*n*sizeof(float);
  uint64_t cbytes = m*n*sizeof(float);

  float *device_a;
  float *device_b;
  float *device_c;
  hipMalloc(&device_a, abytes);
  hipMalloc(&device_b, bbytes);
  hipMalloc(&device_c, cbytes);

  int GRID = ((n*n)+(SIZE-1))/SIZE;
  if(GRID == 0) {
    GRID = 1;
  }
  copy_block<<<GRID, SIZE>>>(device_a, m, k, pack->a, pack->aTop, pack->aLeft, pack->aRowskip);
  // cudaDeviceSynchronize();

  copy_block<<<GRID, SIZE>>>(device_b, k, n, pack->b, pack->bTop, pack->bLeft, pack->bRowskip);
  hipDeviceSynchronize();

  timer_report_tick(&t, "--- memcpy to gpu");

  float alpha = 1.0;
  float beta = 0.0;
  hipblasHandle_t handle;
  hipblasCreate(&handle);  
  hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, device_a, m, device_b, k, &beta, device_c, m);
  hipblasDestroy(handle);
  timer_report_tick(&t, "      hipblasSgemm");


  hipMemcpy(pack->c, device_c, cbytes, hipMemcpyDeviceToHost);

  hipFree(device_a);
  hipFree(device_b);
  hipFree(device_c);
  timer_report_tick(&t, "  memcpy from gpu");

  __atomic_store_n(&(pack->finished), (bool)true, __ATOMIC_SEQ_CST); /* VERY IMPORTANT! */
  return NULL;
}


extern "C" struct fancy_dmm_package * 
fancy_dmm_spawn(
  char * gpu_id,
  int64_t gpu_id_str_len,
  float * a,     // on device
  int64_t aTop,
  int64_t aLeft,
  int64_t aRowskip,
  float * b,     // on device
  int64_t bTop,
  int64_t bLeft,
  int64_t bRowskip,
  float * c,     // on host
  int64_t cTop,
  int64_t cLeft,
  int64_t cRowskip,
  int64_t m,
  int64_t n,
  int64_t k)
{
  struct fancy_dmm_package *pack = (fancy_dmm_package*)malloc(sizeof(struct fancy_dmm_package));

  pack->a = a;
  pack->aTop = aTop;
  pack->aLeft = aLeft;
  pack->aRowskip = aRowskip;

  pack->b = b;
  pack->bTop = bTop;
  pack->bLeft = bLeft;
  pack->bRowskip = bRowskip;

  pack->c = c;
  pack->cTop = cTop;
  pack->cLeft = cLeft;
  pack->cRowskip = cRowskip;

  pack->m = m;
  pack->n = n;
  pack->k = k;

  pack->finished = false;

  set_cuda_device(gpu_id, gpu_id_str_len);
  fancy_dmm_func(pack);

  // if (0 != pthread_create(&(pack->friends), NULL, &fancy_dmm_func, pack)) {
  //   printf("ERROR: glue.c: futdMMSpawn: pthread_create failed\n");
  //   exit(1);
  // }

  return pack;
}


/* TODO: memcpy from GPU back to pack->output
 *
 * (NOTE: futhark_values is equivalent of this memcpy. needs to be replaced) */
extern "C" void fancy_dmm_finish(
  struct fancy_dmm_package * pack)
{
  // if (0 != pthread_join(pack->friends, NULL)) {
  //   printf("ERROR: glue.c: pthread_join failed\n");
  //   exit(1);
  // }

  free(pack);
}


// ===========================================================================


struct fancy_two_dmm_package {
  float * a;  // on device
  int64_t aTop1;
  int64_t aLeft1;
  int64_t aTop2;
  int64_t aLeft2;
  int64_t aRowskip;
  float * b;  // on device
  int64_t bTop1;
  int64_t bLeft1;
  int64_t bTop2;
  int64_t bLeft2;
  int64_t bRowskip;
  float * c;  // on host
  int64_t cTop;
  int64_t cLeft;
  int64_t cRowskip;
  int64_t n;

  /* these should stay */
  bool finished;
  pthread_t friends;
};



extern "C"
void* fancy_two_dmm_func(void* rawArg) {
  struct my_timer_t t;
  timer_begin(&t, "fancy_two_dmm_func");

  struct fancy_two_dmm_package *pack = (struct fancy_two_dmm_package *)rawArg;

  uint64_t n = pack->n;
  uint64_t rowbytes = n*sizeof(float);
  uint64_t bytes = n*rowbytes;


  float *device_c;
  hipMalloc(&device_c, bytes);
  // for (int64_t j = 0; j < n; j++) {
  //   float *host_start = pack->c + (pack->cTop + j) * pack->cRowskip + pack->cLeft;
  //   cudaMemcpyAsync(device_c + j*n, host_start, rowbytes, cudaMemcpyHostToDevice);
  // }

  // cudaDeviceSynchronize();
  // timer_report_tick(&t, "----- memcpy C to gpu");

  float *tmp_a;
  float *tmp_b;
  hipMalloc(&tmp_a, bytes);
  hipMalloc(&tmp_b, bytes);


  int GRID = ((n*n)+(SIZE-1))/SIZE;
  if(GRID == 0) {
    GRID = 1;
  }
  copy_block<<<GRID, SIZE>>>(tmp_a, n, n, pack->a, pack->aTop1, pack->aLeft1, pack->aRowskip);
  copy_block<<<GRID, SIZE>>>(tmp_b, n, n, pack->b, pack->bTop1, pack->bLeft1, pack->bRowskip);
  hipDeviceSynchronize();
  timer_report_tick(&t, "- memcpy A1,B1 on gpu");

  float alpha = 1.0;
  float beta = 0.0;
  hipblasHandle_t handle;
  hipblasCreate(&handle);  
  hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &alpha, tmp_a, n, tmp_b, n, &beta, device_c, n);
  hipblasDestroy(handle);
  timer_report_tick(&t, "   cublasSgemm(A1,B1)");
  

  copy_block<<<GRID, SIZE>>>(tmp_a, n, n, pack->a, pack->aTop2, pack->aLeft2, pack->aRowskip);
  copy_block<<<GRID, SIZE>>>(tmp_b, n, n, pack->b, pack->bTop2, pack->bLeft2, pack->bRowskip);
  hipDeviceSynchronize();
  timer_report_tick(&t, "  memcpy A2,B2 on gpu");


  beta = 1.0;
  hipblasCreate(&handle);  
  hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &alpha, tmp_a, n, tmp_b, n, &beta, device_c, n);
  hipblasDestroy(handle);
  timer_report_tick(&t, "   cublasSgemm(A2,B2)");

  // for (int64_t j = 0; j < n; j++) {
  //   float *host_start = pack->c + (pack->cTop + j) * pack->cRowskip + pack->cLeft;
  //   cudaMemcpyAsync(host_start, device_c + j*n, rowbytes, cudaMemcpyDeviceToHost);
  // }
  // cudaDeviceSynchronize();

  hipMemcpy(pack->c, device_c, bytes, hipMemcpyDeviceToHost);
  hipFree(tmp_a);
  hipFree(tmp_b);
  hipFree(device_c);
  timer_report_tick(&t, "    memcpy C from gpu");

  __atomic_store_n(&(pack->finished), (bool)true, __ATOMIC_SEQ_CST); /* VERY IMPORTANT! */
  return NULL;
}


extern "C" struct fancy_two_dmm_package * 
fancy_two_dmm_spawn(
  char * gpu_id,
  int64_t gpu_id_str_len,
  float * a,     // on device
  int64_t aTop1,
  int64_t aLeft1,
  int64_t aTop2,
  int64_t aLeft2,
  int64_t aRowskip,
  float * b,     // on device
  int64_t bTop1,
  int64_t bLeft1,
  int64_t bTop2,
  int64_t bLeft2,
  int64_t bRowskip,
  float * c,     // on host
  int64_t cTop,
  int64_t cLeft,
  int64_t cRowskip,
  int64_t n)
{
  struct fancy_two_dmm_package *pack = (fancy_two_dmm_package*)malloc(sizeof(struct fancy_two_dmm_package));

  pack->a = a;
  pack->aTop1 = aTop1;
  pack->aLeft1 = aLeft1;
  pack->aTop2 = aTop2;
  pack->aLeft2 = aLeft2;
  pack->aRowskip = aRowskip;

  pack->b = b;
  pack->bTop1 = bTop1;
  pack->bLeft1 = bLeft1;
  pack->bTop2 = bTop2;
  pack->bLeft2 = bLeft2;
  pack->bRowskip = bRowskip;

  pack->c = c;
  pack->cTop = cTop;
  pack->cLeft = cLeft;
  pack->cRowskip = cRowskip;

  pack->n = n;

  pack->finished = false;

  set_cuda_device(gpu_id, gpu_id_str_len);
  fancy_two_dmm_func(pack);

  // if (0 != pthread_create(&(pack->friends), NULL, &fancy_two_dmm_func, pack)) {
  //   printf("ERROR: glue.c: futdMMSpawn: pthread_create failed\n");
  //   exit(1);
  // }

  return pack;
}


// extern "C" uint8_t fancy_two_dmm_poll(struct fancy_two_dmm_package *pack) {
//   return pack->finished ? 1 : 0;
// }


extern "C" void fancy_two_dmm_finish(
  struct fancy_two_dmm_package * pack)
{
  // if (0 != pthread_join(pack->friends, NULL)) {
  //   printf("ERROR: glue.c: pthread_join failed\n");
  //   exit(1);
  // }

  free(pack);
}