#include "hip/hip_runtime.h"
#include "../headers/export.h"
#include "../headers/hofs.h"

//Builtin function pointers
#include "../funcptrs/builtin_tabulate_and_map_float.h"
#include "../funcptrs/builtin_reduce_and_scan_float.h"
#include "../funcptrs/builtin_filter_float.h"

//User defined function pointers
#include "../funcptrs/user_tabulate_float.h"
#include "../funcptrs/user_map_float.h"
#include "../funcptrs/user_reduce_float.h"
#include "../funcptrs/user_scan_float.h"
#include "../funcptrs/user_filter_float.h"
#include "../funcptrs/user_zipwith_float.h"

#include <stdio.h>
#include <time.h>


extern "C"
void * cublasSGEMM(void* A, void* B, void* C, int m, int n, int k, void* f){
    cublasDGEMM_fun hof = (cublasDGEMM_fun)f;
    
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, 1, (float*) A, m, (float*) B, k, 0, (float*) C, m);
    hipblasDestroy(handle);

    return C;
}

__global__ 
void test_cuda(float* arr, int len, test_cuda f){
  
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
    printf("hello from thread %d\n", idx);
}

extern "C"
void* test_cuda(int size, void* f){
  
  test_cuda hof = (test_cuda)f;
  
  void* dev_ptr;
  hipMalloc(&dev_ptr, sizeof(float) * size);

  int blockNum = (size / 256) + 1;
  test_cuda<<<1, 10>>>((float*)dev_ptr, size, hof);
  hipDeviceSynchronize();
  return dev_ptr;

}