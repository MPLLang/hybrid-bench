#include "hip/hip_runtime.h"
#include "../headers/export.h"
#include <stdio.h>

extern "C"
void init_gpu(){
  hipDeviceSynchronize();
}

extern "C"
void* allocate_on_gpu(size_t size, int smltype){
	size_t typesize;
	if(smltype == 0){
		typesize = sizeof(int);
	}
	else{
		typesize = sizeof(float);
	}

	void* ret_ptr;
	hipMalloc(&ret_ptr, typesize * size);
	return ret_ptr;
}

extern "C"
void* copy_float_into_gpu(Pointer src, int size){
	
	void* ret_ptr;
	hipMalloc(&ret_ptr, sizeof(float) * size);
  hipMemcpy(ret_ptr, src, sizeof(float) * size, hipMemcpyHostToDevice);
  
  return ret_ptr;
}

extern "C"
void* copy_int_into_gpu(Pointer src, int size){
	printf("I have entered copy to gpu\n");
	void* ret_ptr;
	hipMalloc(&ret_ptr, sizeof(int) * size);
  hipMemcpy(ret_ptr, src, sizeof(int) * size, hipMemcpyHostToDevice);
  
  return ret_ptr;
}

extern "C"
void copy_float_gpu(Pointer dest, void* gpuarr, size_t size){
	size_t typesize = sizeof(float);
  hipMemcpy(dest, gpuarr, size * typesize, hipMemcpyDeviceToHost);
}

extern "C"
void copy_int_gpu(Pointer dest, void* gpuarr, size_t size){
  printf("i have entered the function\n");
	size_t typesize = sizeof(int);
  int* ptr = (int*)dest;    
  hipMemcpy(ptr, gpuarr, size * typesize, hipMemcpyDeviceToHost);
  printf("this is my printf: %d", ptr[0]);
}

extern "C"
void free_gpu_ptr(void* ptr){
	hipFree(ptr);
}

__global__
void initwith_int(int* arr, int b, int len){
  int idx = threadIdx.x + blockIdx.x * blockDim.x;

  if(idx < len){
    arr[idx] = b;
  }
}
extern "C"
void* initInt_gpu(int size, int b){
  void* dev_ptr;
  hipMalloc(&dev_ptr, sizeof(int) * size);

  int blocks = (size / 256) + 1;
  initwith_int<<<blocks, 256>>>((int*)dev_ptr, b, size);
  hipDeviceSynchronize();
  return dev_ptr;
}

__global__
void initwith_float(float* arr, float b, int len){
  int idx = threadIdx.x + blockIdx.x * blockDim.x;

  if(idx < len){
    arr[idx] = b;
  }
}
extern "C"
void* initFloat_gpu(int size, Real64 b){
  void* dev_ptr;
  hipMalloc(&dev_ptr, sizeof(Real64) * size);

  int blocks = (size / 256) + 1;
  initwith_float<<<blocks, 256>>>((float*)dev_ptr, b, size);
  hipDeviceSynchronize();
  return dev_ptr;
}

extern "C"
void* copy(void* in, int size, int smltype){
	size_t typesize;
	if(smltype == 0){
		typesize = sizeof(int);
	}
	else{
		typesize = sizeof(Real64);
	}

	void* ret_ptr;
	hipMalloc(&ret_ptr, typesize * size);
  hipMemcpy(ret_ptr, in, typesize * size, hipMemcpyDeviceToDevice);
  return ret_ptr;
}

